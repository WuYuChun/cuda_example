
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(int argc ,char *argv[]){

    std::chrono::high_resolution_clock::time_point  start_beig,stop_end;

    int N = 1<<20;
    float *x, *y;
    float *x_device, *y_device;

    //cpu data
    x = (float *)malloc(N*sizeof(float));
    y = (float *)malloc(N*sizeof(float));


    start_beig = std::chrono::high_resolution_clock::now();
    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
    stop_end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_end - start_beig).count();
    std::cout << "hipMalloc set data time : " << duration / 1000.0f << " ms\n";

    //device data
    start_beig = std::chrono::high_resolution_clock::now();
    hipMalloc(&x_device,N*sizeof(float));
    hipMalloc(&y_device,N*sizeof(float));
    hipMemcpy(x_device,x,N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(y_device,y,N*sizeof(float),hipMemcpyHostToDevice);
    stop_end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_end - start_beig).count();
    std::cout << "hipMalloc time : " << duration / 1000.0f << " ms\n";


    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    start_beig = std::chrono::high_resolution_clock::now();

    // Run kernel on 1M elements on the GPU
    add<<<numBlocks, blockSize>>>(N, x_device, y_device);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    hipMemcpy(y,y_device,N*sizeof(float),hipMemcpyDeviceToHost);


    stop_end = std::chrono::high_resolution_clock::now();
    duration = std::chrono::duration_cast<std::chrono::microseconds>(stop_end - start_beig).count();
    std::cout << "hipMalloc exe : " << duration / 1000.0f << " ms\n";

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(y[i]-3.0f));
        if(maxError > 0){
            std::cout << "i: " << i << " y[i]: " << y[i] << std::endl;
        }
    }

    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}